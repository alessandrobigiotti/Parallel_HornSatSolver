/*Created by Alessandro Bigiotti*/

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
// A function to show some GPU Information
int main(){

  // check the number of devices
	int nDevices;
	hipGetDeviceCount(&nDevices);

  // for each device print some informations
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		printf("Device Number: %d\n", i);
		printf(" Info: \n");
		printf("  Device name: %s\n", properties.name);
		printf("  Memory Clock Rate (KHz): %d\n", properties.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", properties.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*properties.memoryClockRate*(properties.memoryBusWidth/8)/1.0e6);
		printf(" Computing Capabilities: \n");
		printf("  Max Threads per Block: %d\n", properties.maxThreadsPerBlock);
		printf("  Max Threads Dim: %d\n", properties.maxThreadsDim[3]);
		printf("  Max GridSize: %d\n", properties.maxGridSize[3]);
		printf("  WarpSize: %d\n", properties.warpSize);
		printf("  Total Global Mem(GB): %f\n", ((properties.totalGlobalMem/1024.0)/1024.0)/1024.0);
		printf("  Shered Mem per Block(MB): %f\n\n", (properties.sharedMemPerBlock/1024.0)/1024);
	}
	return 0;
}
