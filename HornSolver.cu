/*Created by Alessandro Bigiotti*/

// Include C Library
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <error.h>
#include <unistd.h>
#include <string.h>

// Include CUDA Library
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "./otherlib/book.h"

// include dataStructure file
#include "./dataStruct.c"

// include the Kernel function
#include "./kernel_function.cu"

// include an Auxiliary function to set the GPU Memory, GPU Grid/Block, and call the kernel functions
#include "./cuda_setting.cu"

// include an Auxiliary function to read the CNF file,
// Construct the incidence Matrix of the Graph associated to the formula
// Construct the Compact Indicence Matrix of the Graph
// ...
#include "./constructGraph.c"

#define MYCEIL( a, b ) ( (((a) / (b)) + ( (((a) % (b)) == 0) ? 0 : 1 )) )

// The Prototype of the function in cuda_setting.cu
void HornCuda(int *nextpos, int *lett, int *clause, int *tipo, int *poslet, int *num_args, int *matrixelem, int *row, int *col, int n, int m, int nk);

// The Prototype of the function in construct.c
void Construct();

int main()
{

  // Allocate all of the data structures
  Construct();

  // call the function to set the GPU Parameters and call the kernel functions
  HornCuda(nextpos, lett, clause, tipo, poslet, num_args, matrixelem, row, col, n, m, nk);
	
  // free the memory used
  free(matrix);
  free(matrixelem);
  free(col);
  free(row);
  free(lett);
  free(clause);
  free(poslet);
  free(tipo);
  free(nextpos);
	
  return 0;
}
