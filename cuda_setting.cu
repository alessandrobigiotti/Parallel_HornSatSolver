/*Created by Alessandro Bigiotti*/

#define MYCEIL( a, b ) ( (((a) / (b)) + ( (((a) % (b)) == 0) ? 0 : 1 )) )

// This function do:
//    1. set the cuda memory
//    2. calculate and set the cuda Grid/Block dimension
//    3. call the kernel functions
//    4. show the satisfiability
//    5. write in a file Result_Assignments.txt the literal assignments
void HornCuda(int *nextpos, int *lett, int *clause, int *tipo, int *poslet, int *num_args, int *matrixelem, int *row, int *col, int n, int m, int nk)
{

	printf("\nSetting up GPU...\n");
	// Select the GPU Device
	HANDLE_ERROR(hipSetDevice(0));

	// Allocate GPU Memory
	HANDLE_ERROR(hipMalloc((void**)&dev_matrixelem, nk * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_col, nk * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_row, (n + 1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_numargs, m * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_nextpos, (m + 2) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_lett, n * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_clause, m * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_tipo, m * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_poslet, m * sizeof(int)));

	// Copy the data in the Host Memory to Device memory
	HANDLE_ERROR(hipMemcpy(dev_matrixelem, matrixelem, nk * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_col, col, nk * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_row, row, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_numargs, num_args, m * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_nextpos, nextpos, (m + 2) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_lett, lett, n * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_clause, clause, m * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_tipo, tipo, m * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_poslet, poslet, m * sizeof(int), hipMemcpyHostToDevice));

	// Calculate the Block number
	int numblock = MYCEIL(m, 480);

  // Set the Grid and Block Dimension
	dim3 dimGrid(numblock, 1);
	dim3 dimBlock(480, 1, 1);

  // Create an Event to see the execution time
	hipEvent_t start = 0, stop = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int b = numblock;
  int t = 480;

	printf("Calling UnitClause Kernel\n");
	// Call the UnitClause Kernel
	UnitClause << <dimGrid, dimBlock >> >(NValid, b, t, dev_nextpos, dev_numargs, dev_lett, dev_clause, dev_matrixelem, dev_col, dev_row, dev_poslet, dev_tipo, n, m, nk);
  // Read the results after the call
	HANDLE_ERROR(hipMemcpy(nextpos, dev_nextpos, (m + 2) * sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(lett, dev_lett, n*sizeof(int), hipMemcpyDeviceToHost));

  	// Check Satisfiability to continue
	if (nextpos[m] == 1){
		check = true;
	}


  // Iterate until the formula is satisfiable or not
	while (check){
		check = false;
		nextpos[m] = 0;
    // Copy the updated values into GPU Memory
		HANDLE_ERROR(hipMemcpy(dev_lett, lett, n * sizeof(int), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(dev_nextpos, nextpos, (m + 2) * sizeof(int), hipMemcpyHostToDevice));
		printf("Calling Propagate Kernel\n");
    // Call Propagate Kernel
		Propagate<< <dimGrid, dimBlock >> >(NValid, b, t, dev_nextpos, dev_numargs, dev_lett, dev_clause, dev_matrixelem, dev_col, dev_row, dev_poslet, dev_tipo, n, m, nk);
    // Read the results
		HANDLE_ERROR(hipMemcpy(nextpos, dev_nextpos, (m + 2) * sizeof(int), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(lett, dev_lett, n*sizeof(int), hipMemcpyDeviceToHost));
    // Check Satisfiability to continue
		if (nextpos[m] == 1 && nextpos[m + 1] != NValid){
			check = true;
		}
	}

  // Print the relust (Yes or Not)
	if (nextpos[m + 1] == NValid){
		printf("\nNO, the formula is UNSATISFIABLE\n");
	}
	else{
		printf("\nYES, the formula is SATISFIABLE\n");
		// Retrieve the current directory path
	  getcwd(currentDirectory, sizeof(currentDirectory));
	  printf("Directory Corrente: %s\n", currentDirectory);

		// Create a file to write the assignment values
		strcat(strcpy(path_to_assignment, currentDirectory), "/data/Result_Assignments.txt");
		ptr = fopen(path_to_assignment,"w");
		for (int i = 0; i < n; i++){
			fprintf(ptr, "lett: %d, val: %d\n", i, lett[i]);
		}
		fclose(ptr);
	}

	// Stop the event and read the Execution Time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("\n GPU execution time: %f /sec\n", elapsedTime);

	// Free the GPU Memory
	hipFree(dev_matrixelem);
	hipFree(dev_col);
	hipFree(dev_row);
	hipFree(dev_lett);
	hipFree(dev_clause);
	hipFree(dev_poslet);
	hipFree(dev_tipo);
	hipFree(dev_nextpos);
	hipFree(dev_numargs);

	 // Free the GPU Device
	 HANDLE_ERROR(hipDeviceReset());
}
