
#include <hip/hip_runtime.h>
/*Created by Alessandro Bigiotti*/

// Kernel Function to calculate the Unit Clause of the formula
__global__ void UnitClause(int NValid, int b, int t, int *nextpos, int *num_args, int *lett, int *clause, int *matrixelem, int *col, int *row, int *poslet, int *tipo, int n, int m, int nk)
{

	int idx = threadIdx.x;
	int idblock = blockIdx.x;
	int indexpos = 0;
	int indexclause = 0;

	if (idblock*t + idx < m){
		if (tipo[idblock*t + idx] == 1){
			indexpos = poslet[idblock*t + idx];
			indexclause = idblock*t + idx;
			nextpos[indexclause] = 1;
			nextpos[m] = 1;
			lett[indexpos] = 1;
		}
	}
}

// kernel_function to calculate the Propagation of the assignments
__global__ void Propagate(int NValid, int b, int t, int *nextpos, int *num_args, int *lett, int *clause, int *matrixelem, int *col, int *row, int *poslet, int *tipo, int n, int m, int nk)
{
	int idx = threadIdx.x;
	int idblock = blockIdx.x;
	int indexpos = 0;
	int indexnextpos = 0;

	if (idblock*t + idx < m){
		if (nextpos[idblock*t + idx] == 1){
			indexpos = poslet[idblock*t + idx];
			nextpos[idblock*t + idx] = 0;
			for (int i = row[indexpos]; i < row[indexpos + 1]; i++){
				if (matrixelem[i] == 2){
					int old = atomicSub(num_args + col[i], 1);
					printf("%d\n",old);
					if (old == 1){
						indexnextpos = poslet[col[i]];
						if (indexnextpos != NValid){
							if (lett[indexnextpos] == 0){
								lett[indexnextpos] = 1;
								nextpos[col[i]] = 1;
								nextpos[m] = 1;
							}

						}
						else{
							if (tipo[col[i]] % 2 == 0){
								nextpos[m + 1] = NValid;
								break;
							}
						}

					}
				}
			}
		}
	}
}
